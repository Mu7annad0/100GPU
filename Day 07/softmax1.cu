#include <iostream>
#include <hip/hip_runtime.h>


#define CUDA_CHECK(ans)                        \
    {                                          \
        cudaAssert((ans), __FILE__, __LINE__); \
    }
inline void cudaAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA error %s: %s at %s: %d\n",
                hipGetErrorName(code), hipGetErrorString(code),
                file, line);
        exit(code);
    }
}

#define M_PI 3.14159265358979323846f
float random_normal_clamped(float min, float max) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float num = sqrtf(-2.0f * logf(u1)) * cosf(2.0f * M_PI * u2);
    if (num < min)
        return min;
    if (num > max)
        return max;
    return num;
}

__global__ void softmaxKernel(float* __restrict__ A, float* __restrict__ B, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M){
        float m = -1 * INFINITY;
        float L = 0.0f;

        for (int col = 0; col < N; col++){
            m = fmaxf(m, A[row * N + col]);
        }

        for (int col = 0; col < N; col++){
            L += expf(A[row * N + col] - m);
        }

        for (int col = 0; col < N; col++){
            B[row * N + col] = expf(A[row * N + col] - m) / L;
        }
    }
}

int main() {
    int M = 1024;
    int N = 512;

    int size = M*N;

    float* A_h = (float*)malloc(size*sizeof(float));
    float* B_h = (float*)malloc(size*sizeof(float));

    // Initialize input matrix with random values
    for (int i = 0; i < size; i++){
        A_h[i] = random_normal_clamped(-5, 5);
    }

    float *A_d, *B_d;

    int blockSize = 215;
    int gridSize = (M + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms=0.0f;

    // GPU memory allocation
    hipEventRecord(start);
    CUDA_CHECK(hipMalloc(&A_d, size*sizeof(float)));
    CUDA_CHECK(hipMalloc(&B_d, size*sizeof(float)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> GPU allocation time: %f ms\n", ms);

    // Copy data to GPU
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(A_d, A_h, size * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Host to device transfer time: %f ms\n", ms);

    // Run kernel
    hipEventRecord(start);
    softmaxKernel<<<gridSize, blockSize>>>(A_d, B_d, M, N);
    CUDA_CHECK(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Kernel execution time: %f ms\n", ms);

    // Copy results back
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(B_h, B_d, size * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Device to host transfer time: %f ms\n", ms);

    // Free memory
    free(A_h);
    free(B_h);
    hipFree(A_d);
    hipFree(B_d);

    return 0;
}