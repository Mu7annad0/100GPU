#include <stdio.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

#define M_PI 3.14159265358979323846f
float random_normal_clamped(float min, float max) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float num = sqrtf(-2.0f * logf(u1)) * cosf(2.0f * M_PI * u2);
    if (num < min)
        return min;
    if (num > max)
        return max;
    return num;
}

// CUDA kernel for ReLU function
__global__ void reluKernel(float* __restrict__ A, float* __restrict__ B, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M){
        for (int col = 0; col < N; col++){
            B[row * N + col] = fmax(0, A[row * N + col]);
        }
    }
}

int main() {
    int M = 1024;
    int N = 512;

    int size = M*N;

    float* A_h = (float*)malloc(size*sizeof(float));
    float* B_h = (float*)malloc(size*sizeof(float));

    // Initialize input matrix with random values
    for (int i = 0; i < size; i++){
        A_h[i] = random_normal_clamped(-5, 5);
    }

    float *A_d, *B_d;

    int blockSize = 215;
    int gridSize = (M + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms=0.0f;

    // GPU memory allocation
    hipEventRecord(start);
    CUDA_CHECK(hipMalloc(&A_d, size*sizeof(float)));
    CUDA_CHECK(hipMalloc(&B_d, size*sizeof(float)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> GPU allocation time: %f ms\n", ms);

    // Copy data to GPU
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(A_d, A_h, size * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Host to device transfer time: %f ms\n", ms);

    // Run kernel
    hipEventRecord(start);
    reluKernel<<<gridSize, blockSize>>>(A_d, B_d, M, N);
    CUDA_CHECK(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Kernel execution time: %f ms\n", ms);

    // Copy results back
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(B_h, B_d, size * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Device to host transfer time: %f ms\n", ms);

    // Free memory
    free(A_h);
    free(B_h);
    hipFree(A_d);
    hipFree(B_d);

    return 0;
}