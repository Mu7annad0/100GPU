#include <iostream>
#include <hip/hip_runtime.h>


#define CUDA_CHECK(ans)                        \
    {                                          \
        cudaAssert((ans), __FILE__, __LINE__); \
    }
inline void cudaAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA error %s: %s at %s: %d\n",
                hipGetErrorName(code), hipGetErrorString(code),
                file, line);
        exit(code);
    }
}

#define M_PI 3.14159265358979323846f
float random_normal_clamped(float min, float max) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float num = sqrtf(-2.0f * logf(u1)) * cosf(2.0f * M_PI * u2);
    if (num < min)
        return min;
    if (num > max)
        return max;
    return num;
}

__global__ void partialSumKernel(int *inp, int *out, int n) {
    extern __shared__ int sharedMemory[];
    
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x*2 + tid;
    if (index < n) {
        // Load input into shared memory
        sharedMemory[tid] = inp[index]+inp[index+blockDim.x];
        __syncthreads();
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            int temp = 0;
            if (tid >= stride) {
                temp = sharedMemory[tid - stride];
            }
            __syncthreads();
            sharedMemory[tid] += temp;
            __syncthreads();
        }
        // Write result to global memory
        out[index] = sharedMemory[tid];
    }
}


int main() {
    int N = 1024;  // Total number of elements

    // Allocate host memory
    int* inp_h = (int*)malloc(N * sizeof(int));
    int* out_h = (int*)malloc(N * sizeof(int));

    // Initialize input array
    for (int i = 0; i < N; i++) {
        inp_h[i] = i + 1;  // Example initialization
    }

    // Device pointers
    int *inp_d, *out_d;

    int blockSize = 256;  // Typical block size
    int gridSize = (N + 2*blockSize - 1) / (2*blockSize);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    // GPU memory allocation
    hipEventRecord(start);
    CUDA_CHECK(hipMalloc(&inp_d, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&out_d, N * sizeof(int)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> GPU allocation time: %f ms\n", ms);

    // Copy data to GPU
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(inp_d, inp_h, N * sizeof(int), hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Host to device transfer time: %f ms\n", ms);

    // Determine shared memory size
    size_t sharedMemSize = blockSize * sizeof(int);

    // Run kernel
    hipEventRecord(start);
    partialSumKernel<<<gridSize, blockSize, sharedMemSize>>>(inp_d, out_d, N);
    CUDA_CHECK(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Kernel execution time: %f ms\n", ms);

    // Copy results back
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(out_h, out_d, N * sizeof(int), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Device to host transfer time: %f ms\n", ms);

    // Optionally, print some results
    printf("First few output elements:\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", out_h[i]);
    }
    printf("\n");

    // Free memory
    free(inp_h);
    free(out_h);
    hipFree(inp_d);
    hipFree(out_d);

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}