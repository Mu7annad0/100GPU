#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void online_softmax_kernel(float* inp, float* out, int M, int N){
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (row < M){
        float max_val = -1 * INFINITY;
        float norm_val = 0.0f;
        for (int col = 0; col < N; col++){
            float curr = inp[row*N + col];
            if (curr > max_val){
                norm_val = norm_val * expf(max_val - curr);
                max_val = curr;
            }
            norm_val += expf(curr - max_val);
        }

        for (int col = 0; col < N; col++){
            out[row*N + col] = expf(inp[row*N + col] - max_val) / norm_val;
        }
    }
}

void run_online_softmax(float *inp, float *out, int M, int N){
    dim3 ThreadsPerBlock(1024);
    dim3 BlocksPerGrid((M + ThreadsPerBlock.x - 1) / ThreadsPerBlock.x);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.f;

    CUDA_CHECK(hipEventRecord(start));
    online_softmax_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(inp, out, M, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf(">> Kernel execution time: %f ms\n", ms);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}