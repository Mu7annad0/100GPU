#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kernels/naive_softmax.cu"
#include "utils.cuh"


int main(){
    int M = 1024;
    int N = 32768;

    float *inp_h = (float*)malloc(M * N * sizeof(float));
    float *out_h = (float*)malloc(M * N * sizeof(float));

    for (int i = 0; i < M * N; i++){
        inp_h[i] = random_normal_clamped(-1.0f, 1.0f);
    }

    float *inp_d, *out_d;
    CUDA_CHECK(hipMalloc(&inp_d, M * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&out_d, M * N * sizeof(float)));

    CUDA_CHECK(hipMemcpy(inp_d, inp_h, M * N * sizeof(float), hipMemcpyHostToDevice));

    run_naive_softmax(inp_d, out_d, M, N);

    CUDA_CHECK(hipMemcpy(out_h, out_d, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Free memory
    free(inp_h);
    free(out_h);
    hipFree(inp_d);
    hipFree(out_d);

    return 0;
}